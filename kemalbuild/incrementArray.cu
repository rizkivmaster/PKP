#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

void incrementArrayOnHost(float *a, int N)
{
    int i;
    for (i = 0; i < N; i++)
        a[i] = a[i] + 1.f;
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    //if (idx < N)
        a[idx] = 100.0;
}

int main(void)
{
    float *a_h, *b_h; // pointers to host memory
    float *a_d; // pointer to device memory
    int i, errcode, N = 10;
    size_t size = N * sizeof(float);

    // allocate arrays on host
    a_h = (float *) malloc(size);
    b_h = (float *) malloc(size);

    // allocate array on device
    fprintf(stdout, "Allocating array on device...\n");
    errcode = hipMalloc((void **) &a_d, size);
    if (errcode != hipSuccess)
    {
        fprintf(stderr, "Error invoking hipMalloc (ERRCODE %d)\n", errcode);
        return 1;
    }
    else
    {
        fprintf(stdout, "hipMalloc success\n");
    }

    // initialization of host data
    for (i = 0; i < N; i++)
        a_h[i] = (float) i;

    // copy data from host to device
    errcode = hipMemcpy(a_d, a_h, sizeof(float) * N, hipMemcpyHostToDevice);
    if (errcode != hipSuccess)
    {
        fprintf(stderr, "Error invoking hipMemcpy (ERRCODE %d)\n", errcode);
        return 1;
    }
    else
    {
        fprintf(stdout, "hipMemcpy success\n");
    }

    // do calculation on host
    incrementArrayOnHost(a_h, N);

    // do calculation on device:
    // Part 1 of 2. Compute execution configuration
    dim3 grid(1, 1), block(1, N+1);
    // int blockSize = 4;
    // int nBlocks = N/blockSize + (N%blockSize == 0 ? 0 : 1);

    // Part 2 of 2. Call incrementArrayOnDevice kernel
    // incrementArrayOnDevice<<<nBlocks, blockSize>>>(a_d, N);
    fprintf(stdout, "Begin incrementing array on device\n");
    incrementArrayOnDevice<<<grid, block>>>(a_d, N);
    fprintf(stdout, "Finished incrementing array on device\n");

    // Retrieve result from device and store in b_h
    errcode = hipMemcpy(b_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);
    if (errcode != hipSuccess)
    {
        fprintf(stderr, "Error invoking hipMemcpy (ERRCODE %d)\n", errcode);
        return 1;
    }
    else
    {
        fprintf(stdout, "hipMemcpy success\n");
    }

    // check results
    for (i = 0; i < N; i++)
        // assert(a_h[i] == b_h[i]);
        printf("%f %f\n", a_h[i], b_h[i]);

    // cleanup
    free(a_h); free(b_h); hipFree(a_d);
}


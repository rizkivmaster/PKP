#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1500
#define TILE_SIZE 4
#define NANO 1000000000

void checkCudaError(hipError_t errorCode)
{
    if (errorCode != hipSuccess)
    {
        fprintf(stderr, "Error %d\n", errorCode);
        exit(1);
    }
}

float** createSquareMatOnHost(int size)
{
    int i;
    float **mat;

    mat = (float **) malloc(size * sizeof(float *));
    if (!mat)
    {
        fprintf(stderr, "error allocating row memory");
        exit(1);
    }

    mat[0] = (float *) malloc(size * size * sizeof(float));
    if (!mat[0])
    {
        fprintf(stderr, "error allocating col memory");
        exit(1);
    }

    for (i = 1; i < size; i++)
        mat[i] = mat[i-1] + size;

    return mat;
}

void freeSquareMatOnHost(float **mat)
{
    free(mat[0]);
    free(mat);
}

void printSquareMat(float **mat, int size)
{
    int i, j;
    for (i = 0; i < size; i++, printf("\n"))
        for (j = 0; j < size; j++)
            printf(" %f", mat[i][j]);
}

void multiplySquareMatOnHost(float **C, float **A, float **B, int size)
{
    int i, j, k;
    memset(C[0], 0, size * size * sizeof(float));
    for (i = 0; i < size; i++)
        for (j = 0; j < size; j++)
            for (k = 0; k < size; k++)
                C[i][j] += A[i][k] * B[k][j];
}

__global__ void multiplySquareSerializedMatOnDevice(float *C, float *A, float *B, int size)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if (i < size && j < size)
    {
        int k;
        float sum = 0.0;
        for (k = 0; k < size; k++)
            sum += A[i*size+k] * B[k*size+j];
        C[i*size+j] = sum;
    }
}

long long convertToNsec(long long sec, int nsec)
{
    return sec * NANO + nsec;
}

int main(void)
{
    float **ha, **hb, **hc, **hd;   // host data
    float *da, *db, *dc;            // device data
    int i, j;
    int nbytes = N * N * sizeof(float);
    long long elapsedTime;
    struct timespec ts_start, ts_end;

    // allocate memory in host
    ha = createSquareMatOnHost(N);
    hb = createSquareMatOnHost(N);
    hc = createSquareMatOnHost(N);
    hd = createSquareMatOnHost(N);

    // allocate memory in device
    checkCudaError(hipMalloc((void **) &da, nbytes));
    checkCudaError(hipMalloc((void **) &db, nbytes));
    checkCudaError(hipMalloc((void **) &dc, nbytes));

    // initialize all values to zero
    memset(ha[0], 0, nbytes);
    memset(hb[0], 0, nbytes);
    memset(hc[0], 0, nbytes);
    memset(hd[0], 0, nbytes);

    // set values in ha randomly
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            ha[i][j] = rand() % 10;

    // set values in hb randomly
    srand(time(NULL));
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            hb[i][j] = rand() % 10;

    // copy from host to device
    checkCudaError(hipMemcpy(da, ha[0], nbytes, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(db, hb[0], nbytes, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dc, hc[0], nbytes, hipMemcpyHostToDevice));

    // multiply matrix on host
    clock_gettime(CLOCK_MONOTONIC, &ts_start);
    multiplySquareMatOnHost(hd, ha, hb, N);
    clock_gettime(CLOCK_MONOTONIC, &ts_end);

    // compute elapsed time
    elapsedTime = convertToNsec(ts_end.tv_sec, ts_end.tv_nsec) - convertToNsec(ts_start.tv_sec, ts_start.tv_nsec);
    printf("CPU time: %f\n", (float) elapsedTime / NANO);

    // multiply matrix on device
    int gridSize = (N/TILE_SIZE) + (N%TILE_SIZE>0?1:0);
    dim3 grid(gridSize, gridSize), block(TILE_SIZE, TILE_SIZE);
    clock_gettime(CLOCK_MONOTONIC, &ts_start);
    multiplySquareSerializedMatOnDevice<<<grid, block>>>(dc, da, db, N);
    clock_gettime(CLOCK_MONOTONIC, &ts_end);

    // compute elapsed time
    elapsedTime = convertToNsec(ts_end.tv_sec, ts_end.tv_nsec) - convertToNsec(ts_start.tv_sec, ts_start.tv_nsec);
    printf("CUDA time: %f\n", (float) elapsedTime / NANO);

    // copy from device to host
    checkCudaError(hipMemcpy(hc[0], dc, nbytes, hipMemcpyDeviceToHost));

    // assertion
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            assert(hc[i][j] == hd[i][j]);

    freeSquareMatOnHost(ha);
    freeSquareMatOnHost(hb);
    freeSquareMatOnHost(hc);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

#include <stdio.h>
#include <hip/hip_runtime.h>


void checkCudaError(hipError_t errorCode)
{
    if (errorCode != hipSuccess)
        fprintf(stderr, "Error %d\n", errorCode);
}

int main(void)
{
    float *ha, *hb;     // host data
    float *da, *db;     // device data
    int N = 10, nbytes, i;

    nbytes = N * sizeof(float);
    ha = (float *) malloc(nbytes);
    hb = (float *) malloc(nbytes);
    checkCudaError(hipMalloc((void **) &da, nbytes));
    checkCudaError(hipMalloc((void **) &db, nbytes));

    for (i = 0; i < N; i++)
        ha[i] = 100.0 + i;

    checkCudaError(hipMemcpy(da, ha, nbytes, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(db, da, nbytes, hipMemcpyDeviceToDevice));
    checkCudaError(hipMemcpy(hb, db, nbytes, hipMemcpyDeviceToHost));

    for (i = 0; i < N; i++)
        printf("%f %f\n", ha[i], hb[i]);

    free(ha);
    free(hb);
    hipFree(da);
    hipFree(db);

    return 0;
}

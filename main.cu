#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

__global__ void kernel(int * a)
{
	int idx = blockIdx.x*blockDim.x +threadIdx.x;
	a[idx] = 7;
}


__global__ void kernelByThreadId(int * a)
{
	int idx = blockIdx.x*blockDim.x +threadIdx.x;
	a[idx] = threadIdx.x;
}

__global__ void kernelByBlockId(int * a)
{
	int idx = blockIdx.x*blockDim.x +threadIdx.x;
	a[idx] = blockIdx.x;
}



int main(void)
{
	int N = 10;
	int *host_vector;
	int *dev_vector;


	dim3 blockSize(3,3,3);
	dim3 gridSize(1,1);

	//first experiment
	host_vector = (int*) malloc(N*sizeof(int));

	for(int ii = 0; ii < N ;ii++) host_vector[ii] = 0;

	hipMalloc((void**)&dev_vector,N*sizeof(int));

	hipMemcpy(dev_vector,host_vector,N*sizeof(int),hipMemcpyHostToDevice);

	kernel<<<gridSize,blockSize>>>(dev_vector);

	hipMemcpy(host_vector,dev_vector,N*sizeof(int),hipMemcpyDeviceToHost);

	printf("first experiment\n");

	for(int ii = 0; ii < N ;ii++)	printf("%d \n",host_vector[ii]);
	
	free(host_vector);
	hipFree(dev_vector);

	//second experiment
	host_vector = (int*) malloc(N*sizeof(int));

	for(int ii = 0; ii < N ;ii++) host_vector[ii] = 0;

	hipMalloc((void**)&dev_vector,N*sizeof(int));

	hipMemcpy(dev_vector,host_vector,N*sizeof(int),hipMemcpyHostToDevice);

	kernelByBlockId<<<gridSize,blockSize>>>(dev_vector);

	hipMemcpy(host_vector,dev_vector,N*sizeof(int),hipMemcpyDeviceToHost);

	printf("second experiment\n");

	for(int ii = 0; ii < N ;ii++)	printf("%d \n",host_vector[ii]);
	
	free(host_vector);
	hipFree(dev_vector);

	//third experiment
	host_vector = (int*) malloc(N*sizeof(int));

	for(int ii = 0; ii < N ;ii++) host_vector[ii] = 0;
	hipMalloc((void**)&dev_vector,N*sizeof(int));

	hipMemcpy(dev_vector,host_vector,N*sizeof(int),hipMemcpyHostToDevice);

	kernelByThreadId<<<gridSize,blockSize>>>(dev_vector);

	hipMemcpy(host_vector,dev_vector,N*sizeof(int),hipMemcpyDeviceToHost);

	printf("third experiment\n");

	for(int ii = 0; ii < N ;ii++)	printf("%d \n",host_vector[ii]);
	
	free(host_vector);
	hipFree(dev_vector);
}